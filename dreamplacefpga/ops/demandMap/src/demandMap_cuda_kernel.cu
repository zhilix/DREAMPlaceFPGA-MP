#include "hip/hip_runtime.h"
#include "utility/src/utils.cuh"
#include "utility/src/limits.h"
// local dependency
#include "demandMap/src/demand_function.h"

DREAMPLACE_BEGIN_NAMESPACE

/// define compute_demand_function 
template <typename T>
inline __device__ DEFINE_COMPUTE_DEMAND_FUNCTION(T);

template <typename T, typename AtomicOp>
__global__ void __launch_bounds__(1024, 8) computeDemandMap(
        const int *site_type_map, const T *node_size_x, const T *node_size_y,
        const int num_bins_x, const int num_bins_y, 
        const int width, const int height,
        AtomicOp atomicAddOp,
        typename AtomicOp::type *binCapMap0,
        typename AtomicOp::type *binCapMap2,
        typename AtomicOp::type *binCapMap3,
        typename AtomicOp::type *binCapMap4)
{
    T binW = T(width)/T(num_bins_x);
    T binH = T(height)/T(num_bins_y);

    int idx = blockIdx.x * blockDim.z + threadIdx.z;
    if (idx < width*height)
    {
            int rw = int(idx/height);
            int cl = int(idx%height);

        if (site_type_map[idx] == 1)
        {
            T nodeX = node_size_x[1];
            T nodeY = node_size_y[1];
            T col = DREAMPLACE_STD_NAMESPACE::round(cl/nodeY)*nodeY;
            int iLo = int(rw/binW);
            int jLo = int(col/binH);
            int iHi = DREAMPLACE_STD_NAMESPACE::min(int((rw + nodeX)/binW), num_bins_x-1);
            int jHi = DREAMPLACE_STD_NAMESPACE::min(int((col + nodeY)/binH), num_bins_y-1);

            for (int i = iLo + threadIdx.y; i <= iHi; i += blockDim.y)
            {
                T w = compute_demand_function(i, binW, T(rw), nodeX);
                for (int j = jLo + threadIdx.x; j <= jHi; j += blockDim.x)
                {
                    T h = compute_demand_function(j, binH, col, nodeY);
                    T area = w * h;
                    atomicAddOp(&binCapMap0[i*num_bins_y + j], area);
                }
            }
        } else if (site_type_map[idx] == 2)
        {
            T nodeX = node_size_x[2];
            T nodeY = node_size_y[2];
            T col = DREAMPLACE_STD_NAMESPACE::round(cl/nodeY)*nodeY;
            int iLo = int(rw/binW);
            int jLo = int(col/binH);
            int iHi = DREAMPLACE_STD_NAMESPACE::min(int((rw + nodeX)/binW), num_bins_x-1);
            int jHi = DREAMPLACE_STD_NAMESPACE::min(int((col + nodeY)/binH), num_bins_y-1);

            for (int i = iLo + threadIdx.y; i <= iHi; i += blockDim.y)
            {
                T w = compute_demand_function(i, binW, T(rw), nodeX);
                for (int j = jLo + threadIdx.x; j <= jHi; j += blockDim.x)
                {
                    T h = compute_demand_function(j, binH, col, nodeY);
                    T area = w * h;
                    atomicAddOp(&binCapMap2[i*num_bins_y + j], area);
                }
            }
        } else if (site_type_map[idx] == 3)
        {
            T nodeX = node_size_x[3];
            T nodeY = node_size_y[3];
            T col = DREAMPLACE_STD_NAMESPACE::round(cl/nodeY)*nodeY;
            int iLo = int(rw/binW);
            int jLo = int(col/binH);
            int iHi = DREAMPLACE_STD_NAMESPACE::min(int((rw + nodeX)/binW), num_bins_x-1);
            int jHi = DREAMPLACE_STD_NAMESPACE::min(int((col + nodeY)/binH), num_bins_y-1);
            for (int i = iLo + threadIdx.y; i <= iHi; i += blockDim.y)
            {
                T w = compute_demand_function(i, binW, T(rw), nodeX);
                for (int j = jLo + threadIdx.x; j <= jHi; j += blockDim.x)
                {
                    T h = compute_demand_function(j, binH, col, nodeY);
                    T area = w * h;
                    atomicAddOp(&binCapMap3[i*num_bins_y + j], area);
                }
            }
        } else if (site_type_map[idx] == 4)
        {
            T nodeX = node_size_x[4];
            T nodeY = node_size_y[4];
            T col = DREAMPLACE_STD_NAMESPACE::round(cl/nodeY)*nodeY;
            int iLo = int(rw/binW);
            int jLo = int(col/binH);
            int iHi = DREAMPLACE_STD_NAMESPACE::min(int((rw + nodeX)/binW), num_bins_x-1);
            int jHi = DREAMPLACE_STD_NAMESPACE::min(int((col + nodeY)/binH), num_bins_y-1);
            for (int i = iLo + threadIdx.y; i <= iHi; i += blockDim.y)
            {
                T w = compute_demand_function(i, binW, T(rw), nodeX);
                for (int j = jLo + threadIdx.x; j <= jHi; j += blockDim.x)
                {
                    T h = compute_demand_function(j, binH, col, nodeY);
                    T area = w * h;
                    atomicAddOp(&binCapMap4[i*num_bins_y + j], area);
                }
            }
        }
    }
}



template <typename T, typename AtomicOp>
int computeDemandMapCallKernel(
        const int *site_type_map, const T *node_size_x,
        const T *node_size_y, const int num_bins_x,
        const int num_bins_y, const int width, const int height,
        AtomicOp atomicAddOp,
        typename AtomicOp::type *binCapMap0,
        typename AtomicOp::type *binCapMap2,
        typename AtomicOp::type *binCapMap3,
        typename AtomicOp::type *binCapMap4
        )
{
  int thread_count = 64;
  dim3 blockSize(2, 2, thread_count);

  int block_count = (width*height - 1 + thread_count) / thread_count;

    computeDemandMap<<<block_count, blockSize>>>(
            site_type_map, node_size_x, node_size_y,
            num_bins_x, num_bins_y, width, height,
            atomicAddOp, binCapMap0,
            binCapMap2, binCapMap3, binCapMap4);

    return 0;
}


template <typename T>
int computeDemandMapCudaLauncher(
        const int *site_type_map,
        const T *node_size_x, 
        const T *node_size_y, 
        const int num_bins_x, 
        const int num_bins_y, 
        const int width, 
        const int height, 
        const int deterministic_flag,
        T *binCapMap0,
        T *binCapMap2,
        T *binCapMap3,
        T *binCapMap4
        )
{
    if (deterministic_flag == 1)
    {
    // total die area
    double diearea = width * height;
    int integer_bits = max((int)ceil(log2(diearea)) + 1, 32);
    int fraction_bits = max(64 - integer_bits, 0);
    unsigned long long int scale_factor = (1UL << fraction_bits);
    int num_bins = num_bins_x * num_bins_y;

    unsigned long long int *bin_cap_map_0 = NULL;
    allocateCUDA(bin_cap_map_0, num_bins, unsigned long long int);
    unsigned long long int *bin_cap_map_2 = NULL;
    allocateCUDA(bin_cap_map_2, num_bins, unsigned long long int);
    unsigned long long int *bin_cap_map_3 = NULL;
    allocateCUDA(bin_cap_map_3, num_bins, unsigned long long int);
    unsigned long long int *bin_cap_map_4 = NULL;
    allocateCUDA(bin_cap_map_4, num_bins, unsigned long long int);

    AtomicAddCUDA<unsigned long long int> atomicAddOp(scale_factor);
    int thread_count = 512;

    copyScaleArray<<<(num_bins + thread_count - 1) / thread_count,
                     thread_count>>>(
        bin_cap_map_0, binCapMap0, scale_factor, num_bins);
    copyScaleArray<<<(num_bins + thread_count - 1) / thread_count,
                     thread_count>>>(
        bin_cap_map_2, binCapMap2, scale_factor, num_bins);
    copyScaleArray<<<(num_bins + thread_count - 1) / thread_count,
                     thread_count>>>(
        bin_cap_map_3, binCapMap3, scale_factor, num_bins);
    copyScaleArray<<<(num_bins + thread_count - 1) / thread_count,
                     thread_count>>>(
        bin_cap_map_4, binCapMap4, scale_factor, num_bins);

    computeDemandMapCallKernel<T, decltype(atomicAddOp)>(
                site_type_map, node_size_x, node_size_y,
                num_bins_x, num_bins_y, width, height,
                atomicAddOp, bin_cap_map_0, bin_cap_map_2, bin_cap_map_3, bin_cap_map_4);

    copyScaleArray<<<(num_bins + thread_count - 1) / thread_count,
                     thread_count>>>(binCapMap0,
                     bin_cap_map_0, T(1.0 / scale_factor), num_bins);
    copyScaleArray<<<(num_bins + thread_count - 1) / thread_count,
                     thread_count>>>(binCapMap2,
                     bin_cap_map_2, T(1.0 / scale_factor), num_bins);
    copyScaleArray<<<(num_bins + thread_count - 1) / thread_count,
                     thread_count>>>(binCapMap3,
                     bin_cap_map_3, T(1.0 / scale_factor), num_bins);
    copyScaleArray<<<(num_bins + thread_count - 1) / thread_count,
                     thread_count>>>(binCapMap4,
                     bin_cap_map_4, T(1.0 / scale_factor), num_bins);

    destroyCUDA(bin_cap_map_0);
    destroyCUDA(bin_cap_map_2);
    destroyCUDA(bin_cap_map_3);
    destroyCUDA(bin_cap_map_4);
  } else
    {
        AtomicAddCUDA<T> atomicAddOp;

        computeDemandMapCallKernel<T, decltype(atomicAddOp)>(
                site_type_map, node_size_x, node_size_y,
                num_bins_x, num_bins_y, width, height,
                atomicAddOp, binCapMap0, binCapMap2, binCapMap3, binCapMap4);
    }
    return 0;
}

// manually instantiate the template function
#define REGISTER_KERNEL_LAUNCHER(T)                         \
    template int computeDemandMapCudaLauncher<T>(           \
        const int *site_type_map, const T *node_size_x,     \
        const T *node_size_y, const int num_bins_x,         \
        const int num_bins_y, const int width,              \
        const int height, const int deterministic_flag,     \
        T *binCapMap0, T *binCapMap2, T *binCapMap3, T *binCapMap4);

REGISTER_KERNEL_LAUNCHER(float);
REGISTER_KERNEL_LAUNCHER(double);

DREAMPLACE_END_NAMESPACE
